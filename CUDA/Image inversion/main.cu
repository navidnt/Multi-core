
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <fstream>
//#include <opencv/core.hpp>
#define Max_Size 99532800
#define Block_Size 16

using namespace std;





__global__ void inverter(int* g_idata, int* g_odata, int n){
    int tid = threadIdx.x;
    //g_odata[tid] = g_idata[tid];
    int st = tid * 1092 * 64;
    int ed = st + 1092 * 64;
    for(int i = st; i < ed; i++){
        g_odata[i] = 255 - g_idata[i];
        __syncthreads();
    }
}





//int ph[5000][8000][3];

int main(){
    int cnt = 0;
    int cntRow = 0;
    int cntCol = 0;
    int cntC = 0;
    ifstream inFile;
    ofstream outFile;
    inFile.open("imageMat2.txt");
    outFile.open("imageMat3.txt");
    int *ph_host = new int [Max_Size];
    while(cnt < 7680*4320*3){
        int z;
        inFile >> z;
        //ph[cntRow][cntCol][cntC] = z;
        ph_host[cnt] = z;
        cnt++;
        //outFile << ph[cntRow][cntCol][cntC] << "\n";
        /*if(cntC < 2){
            cntC++;
            continue;
        }
        cntC = 0;
        cntCol ++;
        if(cntCol == 7680){
            cntCol = 0;
            cntRow++;
        }*/
    }

    ///Device var define
    hipEvent_t start, stop;
    hipEventCreate (&start);
    hipEventCreate (&stop);
    int* ph_dev;
    int* nph_dev;
    int* nph_host = new int [Max_Size];
    int SizeOfMax = Max_Size * sizeof(int);
    hipMalloc((void **) &ph_dev, SizeOfMax);
    hipMalloc((void **) &nph_dev, SizeOfMax);

    hipMemcpy((void *)ph_dev, ph_host, SizeOfMax, hipMemcpyHostToDevice);

    hipEventRecord(start);

    inverter <<<1, Block_Size>>> (ph_dev, nph_dev, Max_Size);

    hipEventRecord(stop);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
            if(error!=hipSuccess)
            {
              fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
              exit(-1);
            }
    hipMemcpy((void *) nph_host, nph_dev, SizeOfMax, hipMemcpyDeviceToHost);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);

    cout << nph_host[0] << endl << ms << " ms" << endl << sizeof(float) << endl;

    for(int i = 0; i < Max_Size; i++){
        outFile << nph_host[i] << "\n";
    }

    outFile.close();
    inFile.close();

    hipFree(ph_dev);
    hipFree(nph_dev);
    free(ph_host);
    free(nph_host);
    return 0;
}
