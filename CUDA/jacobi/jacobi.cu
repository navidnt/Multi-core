
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <omp.h>


#define Max_Size 128
#define Block_Size 32

using namespace std;


__global__ void operate(double* g_i, double* g_o, int wid){
    int rowSt =  threadIdx.y * Max_Size/Block_Size;
    int colSt =  threadIdx.x * Max_Size/Block_Size;
    int rowEd = (threadIdx.y + 1) * Max_Size/Block_Size;
    int colEd = (threadIdx.x + 1) * Max_Size/Block_Size;
    double sum = 0;
    for(int row = rowSt; row < rowEd; row++){
      for(int col = colSt; col < colEd; col++){
        if(row > 0 && row < Max_Size-1 && col > 0 && col < Max_Size-1){
          sum += g_i[row*wid+col-1];
          sum += g_i[row*wid+col-wid];
          sum += g_i[row*wid+col+1];
          sum += g_i[row*wid+col+wid];
          g_o[row*wid+col] = double (sum/4);
          sum = 0;
        }
        else{
          g_o[row*wid+col] = g_i[row*wid+col];
        }
      }
    }
}



int main(int argc, char *argv[]) {

    //struct timeval startTime, stopTime;

    int m;
    int n;
    double tol; // = 0.0001;
    //long totalTime;

    m = atoi(argv[1]);
    n = atoi(argv[2]);
    tol = atof(argv[3]);
    float ms;
    //double t[m + 2][n + 2], tnew[m + 1][n + 1], diff, diffmax;
    double *t_host = new double[Max_Size*Max_Size];
    double *tnew_host = new double[Max_Size*Max_Size];
    double diff, diffmax;
    //for (int z = 0; z < 11; z++) {
        //gettimeofday(&startTime, NULL);


        // initialise temperature array
        for (int i = 0; i < m + 2; i++){
            for (int j = 0; j < n + 2; j++){
                t_host[i*(n+2)+j] = 30.0;
                //printf("%d %d thread: %d\n", i, j, omp_get_thread_num());
            }
        }


        {
            // fix boundary conditions
            for (int i = 1; i <= m; i++) {
                t_host[i*(n+2)] = 10.0;
                t_host[(i+1)*(n + 1)+i] = 140.0;
            }
            for (int j = 1; j <= n; j++) {
                t_host[j] = 20.0;
                t_host[(m + 1)*(n+2)+j] = 100.0;
            }
        }

        // main loop
        diffmax = 1000000.0;
        int lev = 0;
        while (diffmax > tol) {
            lev++;
            // update temperature for next iteration
            /*for (int i = 1; i <= m; i++)
                for (int j = 1; j <= n; j++)
                    tnew[i][j] =
                        (t[i - 1][j] + t[i + 1][j] + t[i][j - 1] + t[i][j + 1]) / 4.0;
            */
            //cout << tnew_host[0] << " Hello\n";
            hipEvent_t start, stop;
            hipEventCreate (&start);
            hipEventCreate (&stop);
            double *t_dev, *tnew_dev;
            int SizeOfMax = Max_Size * Max_Size * sizeof(double);
            hipMalloc((void **) &t_dev, SizeOfMax);
            hipMalloc((void **) &tnew_dev, SizeOfMax);
            hipMemcpy(t_dev, t_host, SizeOfMax, hipMemcpyHostToDevice);
            //int ggg = (Max_Size-1)/Block_Size+1;
            //cout << " SFASF " << ggg << endl;
            //dim3 dimGrid(ggg,ggg);
            dim3 dimBlock(Block_Size,Block_Size);
            hipEventRecord(start);
            operate <<<1, dimBlock>>> (t_dev, tnew_dev, Max_Size);
            hipEventRecord(stop);
            hipDeviceSynchronize();
            hipError_t error = hipGetLastError();
            if(error!=hipSuccess)
            {
              fprintf(stderr,"ERROR: %s\n", hipGetErrorString(error) );
              exit(-1);
            }
            hipMemcpy(tnew_host, tnew_dev, SizeOfMax, hipMemcpyDeviceToHost);
            ms = 0;
            hipEventElapsedTime(&ms, start, stop);
            diffmax = 0.0;
            // work out maximum difference between old and new temperatures
            for (int i = 1; i <= m; i++) {
                for (int j = 1; j <= n; j++) {
                  diff = fabs(tnew_host[i*(n+2)+j] - t_host[i*(n+2)+j]);
                  if(diff > diffmax) {
                    diffmax = diff;
                  }
                  // copy new to old temperatures
                  t_host[i*(n+2)+j] = tnew_host[i*(n+2)+j];
                }
            }
            //cout << "sss " << tnew_host[344] << endl;
            hipFree(t_dev);
            hipFree(tnew_dev);
        }

        //gettimeofday(&stopTime, NULL);
        //totalTime = (stopTime.tv_sec * 1000000 + stopTime.tv_usec) -
           //         (startTime.tv_sec * 1000000 + startTime.tv_usec);

        //printf("%ld\n", totalTime);
        cout << lev << " ms:" << ms << " " << diffmax << endl;
    //}
}


